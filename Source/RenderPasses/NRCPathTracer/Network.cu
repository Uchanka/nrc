#include "hip/hip_runtime.h"
#ifndef __NVCC__        // defined by nvcc complier 
#define __NVCC__
#endif

#include "Network.h"

#include <tiny-cuda-nn/misc_kernels.h>
#include <tiny-cuda-nn/config.h>
#include <tiny-cuda-nn/common.h>

using namespace tcnn;
using precision_t = tcnn::network_precision_t;

#define GPUMatrix GPUMatrix<float, CM>
//typedef tcnn::GPUMatrix<float, RM> GPUMatrix;
//using GPUMatrix = tcnn::GPUMatrix<float, RM>

namespace {

    // 4 steps each frame, with 16384 samples per batch
    unsigned int resolution = 1920 * 1080;    // is a multiple of 256
    const unsigned int batch_size = 1 << 14;
    const unsigned int self_query_batch_size = 1 << 16;     // ~ 57600
    const unsigned int input_dim = 5;         // pos, dir
    const unsigned int output_dim = 3;        // RGB
    //const unsigned int alignment = 16;        // input dim alignment
    const std::string config_path = "../RenderPasses/NRCPathTracer/Data/default_nrc.json";

    // cuda related
    hipStream_t inference_stream;
    hipStream_t training_stream;

    struct _Network { 
        std::shared_ptr<Loss<precision_t>> loss = nullptr;
        std::shared_ptr<Optimizer<precision_t>> optimizer = nullptr;
        std::shared_ptr<NetworkWithInputEncoding<precision_t>> network = nullptr;
        std::shared_ptr<Trainer<float, precision_t, precision_t>> trainer = nullptr;
        //std::shared_ptr<Network<precision_t>> network = nullptr;
        //std::shared_ptr<Encoding<precision_t>> encoding = nullptr;
    };

    struct _Memory {
        // the GPUMatrix class supports MxN matrices only
        // the GPUMatrix store in a continuous area in memory, either row major or column major
        GPUMatrix* training_data = nullptr;
        GPUMatrix* training_target = nullptr;
        GPUMatrix* inference_data = nullptr;
        GPUMatrix* inference_target = nullptr;
        GPUMatrix* training_self_query = nullptr;
        GPUMatrix* training_self_pred = nullptr;
    };

    _Memory* mMemory;
    _Network* mNetwork;
}

// device code helper functions
template <typename T = float3>
__device__ T vec3_mult(T a, T b) {
    return { a.x * b.x, a.y * b.y, a.z * b.z };
}

template <typename T = float3>
__device__ T vec3_add(T a, T b) {
    return { a.x + b.x, a.y + b.y, a.z + b.z };
}

// linear kernels with only x-dim not 1. must be called using linear_kernal()
// blockDim = 128, threadIdx is the index of a thread within a thread block, i.e. in [0, 128)
// reference linear_kernel() for details.
// stride: input dim
template <uint32_t stride, typename T = float>
__global__ void generateBatchSequential(uint32_t n_elements, uint32_t offset, 
    NRC::RadianceQuery* __restrict__ queries, T* __restrict__ data) {
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i + offset < n_elements) {
        uint32_t data_index = i * stride, query_index = i + offset;

        data[data_index + 0] = (T)queries[query_index].pos.x;
        data[data_index + 1] = (T)queries[query_index].pos.y;
        data[data_index + 2] = (T)queries[query_index].pos.z;
        data[data_index + 3] = (T)queries[query_index].dir.x;
        data[data_index + 4] = (T)queries[query_index].dir.y;
    }
}

template <typename T = float>
__global__ void generateTrainingDataFromSamples(uint32_t n_elements, uint32_t offset,
    NRC::RadianceSample* __restrict__ samples, T* __restrict__ self_query_pred,
    T* __restrict__ training_data, T* __restrict__ training_target,
    uint32_t* training_sample_counter, uint32_t* self_query_counter) {
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i + offset > n_elements) return;
    uint32_t data_index = i * 5, sample_index = i + offset;
    uint32_t pred_index = samples[sample_index].idx >= 0 ? samples[sample_index].idx : 0;

    if (sample_index < *training_sample_counter && pred_index < *self_query_counter) {
        float3 factor = samples[sample_index].a, bias = samples[sample_index].b;
        uint32_t output_index = i * 3;

        training_data[data_index + 0] = (T)samples[sample_index].query.pos.x;
        training_data[data_index + 1] = (T)samples[sample_index].query.pos.y;
        training_data[data_index + 2] = (T)samples[sample_index].query.pos.z;
        training_data[data_index + 3] = (T)samples[sample_index].query.dir.x;
        training_data[data_index + 4] = (T)samples[sample_index].query.dir.y;

        float3 pred_radiance = { self_query_pred[pred_index], self_query_pred[pred_index + 1], self_query_pred[pred_index + 2] };
        float3 radiance = vec3_add(vec3_mult(pred_radiance, factor), bias);
        training_target[output_index + 0] = (T)radiance.x;
        training_target[output_index + 1] = (T)radiance.y;
        training_target[output_index + 2] = (T)radiance.z;
    }
}

template <typename T = float>
__global__ void mapPredRadianceToScreen(uint32_t n_elements, uint32_t width,
    T* __restrict__ data, hipSurfaceObject_t output) {
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int x = i % width, y = i / width;
    unsigned int index = i * 3;
    float4 radiance = { data[index + 0] , data[index + 1], data[index + 2], 1.f };
    surf2Dwrite(radiance, output, (int)sizeof(float4) * x, y);
}

template <class T>
__global__ void mapPredRadianceToScreen2(T* __restrict__ data, hipSurfaceObject_t output,
    unsigned int width, unsigned int height) {
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height) {
        unsigned int index = (y * width + x) * 3;
        float4 radiance = { data[index + 0], data[index + 1], data[index + 2], 1.f };
    
//         float greyScale = ((float)x / width) * ((float)y / height);
//        float4 radiance = { greyScale, greyScale, greyScale, 1.f };
        surf2Dwrite(radiance, output, (int)sizeof(float4) * x, y);
    }
}

template <typename T = float>
__global__ void chkNaN(uint32_t n_elements, T* __restrict__ data) {
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i > n_elements) return;
    if (isnan(data[i]) || isinf(data[i])) {
        data[i] = (T)0.f;
    }
}

namespace NRC {
    NRCNetwork::NRCNetwork()
    {
        CUDA_CHECK_THROW(hipStreamCreate(&inference_stream));
        CUDA_CHECK_THROW(hipStreamCreate(&training_stream));

        initializeNetwork();
    }

    NRCNetwork::~NRCNetwork()
    {
        delete mNetwork;
        delete mMemory;
    }

    void NRCNetwork::initializeNetwork()
    {
        mNetwork = new _Network();
        mMemory = new _Memory();

        //initialize network
        std::ifstream f(config_path);
        tcnn::json config = tcnn::json::parse(f, nullptr, true, true);

        json loss_opts = config.value("loss", json::object());
        json optimizer_opts = config.value("optimizer", json::object());
        json network_opts = config.value("network", json::object());
        json encoding_opts = config.value("encoding", json::object());

        mNetwork->loss = std::shared_ptr<Loss<precision_t>>(create_loss<precision_t>(loss_opts) );
        mNetwork->optimizer = std::shared_ptr<Optimizer<precision_t>>(create_optimizer<precision_t>(optimizer_opts));
        mNetwork->network = std::make_shared<NetworkWithInputEncoding<precision_t>>(input_dim, 0, output_dim, encoding_opts, network_opts);
        mNetwork->trainer = std::make_shared<Trainer<float, precision_t, precision_t>>(mNetwork->network, mNetwork->optimizer, mNetwork->loss);

        mMemory->training_data = new GPUMatrix(input_dim, batch_size);
        mMemory->training_target = new GPUMatrix(output_dim, batch_size);
        mMemory->inference_data = new GPUMatrix(input_dim, resolution);
        mMemory->inference_target = new GPUMatrix(output_dim, resolution);
        mMemory->training_self_query = new GPUMatrix(input_dim, self_query_batch_size);
        mMemory->training_self_pred = new GPUMatrix(output_dim, self_query_batch_size);
    }

    void NRCNetwork::reset()
    {
        hipStreamSynchronize(training_stream);
        hipStreamSynchronize(inference_stream);
        mNetwork->trainer->initialize_params(seed);
    }

    void NRCNetwork::inference(RadianceQuery* queries, hipSurfaceObject_t output,
        unsigned int width, unsigned int height)
    {

        unsigned int n_elements = width * height;
        //int n_batches = div_round_up(n_elements, batch_size);
        //int n_queries = next_multiple(n_elements, 256u);
        
        //std::cout << "Inference batch size: " << mMemory->inference_data->rows() << mMemory->inference_data->cols() << std::endl;

        // this input generation process takes about ~1ms.
        linear_kernel(generateBatchSequential<input_dim>, 0, inference_stream, n_elements,
            0, queries, mMemory->inference_data->data());
        
        mNetwork->network->inference(inference_stream, *mMemory->inference_data, *mMemory->inference_target);

        //linear_kernel(mapPredRadianceToScreen<float>, 0, inference_stream, n_elements, width, mMemory->inference_target->data(), output);

        dim3 dimBlock(16, 16), dimGrid(div_round_up(width, 16u), div_round_up(height, 16u));
        mapPredRadianceToScreen2<float> <<<dimGrid, dimBlock, 0, inference_stream >>>
            (mMemory->inference_target->data(), output, width, height);

        hipStreamSynchronize(inference_stream);
    }

    void NRCNetwork::train(RadianceQuery* self_queries, uint32_t* self_query_counter,
        RadianceSample* training_samples, uint32_t* training_sample_counter, float& loss)
    {
        // self query
        linear_kernel(generateBatchSequential<input_dim>, 0, training_stream, self_query_batch_size,
            0, self_queries, mMemory->training_self_query->data());

        mNetwork->network->inference(training_stream, *mMemory->training_self_query, *mMemory->training_self_pred);

        // training
        linear_kernel(generateTrainingDataFromSamples<float>, 0, training_stream, batch_size,
            0, training_samples, mMemory->training_self_pred->data(),
            mMemory->training_data->data(), mMemory->training_target->data(),
            training_sample_counter, self_query_counter);
        linear_kernel(chkNaN<float>, 0, training_stream, mMemory->training_data->n_elements(), mMemory->training_data->data());
        linear_kernel(chkNaN<float>, 0, training_stream, mMemory->training_target->n_elements(), mMemory->training_target->data());
        mNetwork->trainer->training_step(training_stream, *mMemory->training_data, *mMemory->training_target, &loss);
        hipStreamSynchronize(training_stream);
        std::cout << "Loss at current step: " << loss << std::endl;
    }
}
